#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "kernel.cuh"
#define DIFF_T (0.1f)
#define EPS (1.0f)

__global__ void compute_acc(float3 * positionsGPU, float3 * velocitiesGPU, float3 * accelerationsGPU, float* massesGPU, int n_particles)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	for (int j = i+1; j < n_particles; j++)
	{
		float3 diff;
		diff.x = positionsGPU[j].x - positionsGPU[i].x;
		diff.y = positionsGPU[j].y - positionsGPU[i].y;
		diff.z = positionsGPU[j].z - positionsGPU[i].z;

		float dij = diff.x * diff.x + diff.y * diff.y + diff.z * diff.z;

		if (dij < 1.f)
		{
			dij = 10.f;
		}
		else
		{
			dij = rsqrt(dij);
			dij = 10.f * (dij * dij * dij);
		}

		accelerationsGPU[i].x += diff.x * dij * massesGPU[j];
		accelerationsGPU[i].y += diff.y * dij * massesGPU[j];
		accelerationsGPU[i].z += diff.z * dij * massesGPU[j];
		accelerationsGPU[j].x -= diff.x * dij * massesGPU[i];
		accelerationsGPU[j].y -= diff.y * dij * massesGPU[i];
		accelerationsGPU[j].z -= diff.z * dij * massesGPU[i];
	}
}

__global__ void maj_pos(float3 * positionsGPU, float3 * velocitiesGPU, float3 * accelerationsGPU)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	velocitiesGPU[i].x += accelerationsGPU[i].x * 2.f;
	velocitiesGPU[i].y += accelerationsGPU[i].y * 2.f;
	velocitiesGPU[i].z += accelerationsGPU[i].z * 2.f;
	positionsGPU[i].x  += velocitiesGPU[i].x * DIFF_T;
	positionsGPU[i].y  += velocitiesGPU[i].y * DIFF_T;
	positionsGPU[i].z  += velocitiesGPU[i].z * DIFF_T;
}

void update_position_cu(float3* positionsGPU, float3* velocitiesGPU, float3* accelerationsGPU, float* massesGPU, int n_particles)
{
	int nthreads = 128;
	int nblocks =  (n_particles + (nthreads -1)) / nthreads;

	compute_acc<<<nblocks, nthreads>>>(positionsGPU, velocitiesGPU, accelerationsGPU, massesGPU, n_particles);
	maj_pos    <<<nblocks, nthreads>>>(positionsGPU, velocitiesGPU, accelerationsGPU);
}
